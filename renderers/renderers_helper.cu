#include "hip/hip_runtime.h"
#define SWAP(x, y) do {         \
    decltype(x) _x = x;        \
    decltype(y) _y = y;        \
    x = _y;                    \
    y = _x;                    \
} while (0)
#include "renderers_helper.cuh"
#include <stdlib.h>


__host__ __device__ HitObj find_intersection(float ray_x, float ray_y, Spheres spheres, int n, float3 camera_pos)
{
	HitObj res;
	res.x = 0;
	res.y = 0;
	res.z = FLT_MAX;
	res.index = -1;
	float ray_z = -1;
	float radius;
	float3 A = make_float3(ray_x, ray_y, 0);
	// Now hardcode the camera position as 0,0,-500
	float3 B = normalize(A - camera_pos);
	float3 C;
	// I have unit vector for B, so a is 1
	float a = 1.0f;
	float b;
	float c;
	float d;
	float step1;
	float step2;
	for (int i = 0; i < n; i++)
	{
		C = make_float3(spheres.x[i], spheres.y[i], spheres.z[i]);
		//if (ray_x == -199 && ray_y == 200)
		//	printf("%f %f %f\n%f %f %f\n %f %f %f\n", A.x, A.y, A.z, B.x, B.y, B.z, C.x, C.y, C.z);
		radius = spheres.radius[i];
		float3 A_C = A - C;
		b = 2 * dot(B, A_C);
		float tmp = dot(A_C, A_C);
		c = dot(A_C, A_C) - radius * radius;
		d = b * b - 4 * c;

		//if (ray_x == -199 && ray_y == 200)
		//	printf("%f %f %f %f\n", b, c, d, tmp);
		if (d >= 0)
		{
			step1 = (-b - sqrt(d)) / 2;
			step2 = (-b + sqrt(d)) / 2;
			// whole wphere is behind camera
			if (step1 >= 0 && step1 < res.z)
			{
				res.z = step1;
				res.index = i;
			}
			// camera is inside a sphere
			else if (step1 < 0 && step2 >= 0)
			{
				res.z = FLT_MAX;
				res.index = -1;
				break;
			}
		}
	}
	return res;
}





__device__ HitObj find_intersection_gpu_ver2(float ray_x, float ray_y, float* arr, int n, float3 camera_pos, int num)
{
	HitObj res;
	res.x = 0;
	res.y = 0;
	res.z = FLT_MAX;
	res.index = -1;
	float ray_z = -1;
	float radius;
	float3 A = make_float3(ray_x, ray_y, 0);
	// Now hardcode the camera position as 0,0,-500
	float3 B = normalize(A - camera_pos);
	float3 C;
	// I have unit vector for B, so a is 1
	float a = 1.0f;
	float b;
	float c;
	float d;
	int i = 0;
	//int dtmp = num % n;
	while (i < n)
	{
		int offset = i * 4;
		C = make_float3(arr[offset], arr[offset + 1], arr[offset + 2]);
		radius = arr[offset + 3];
		float3 A_C = A - C;
		b = 2 * dot(B, A_C);
		c = dot(A_C, A_C) - radius * radius;
		d = b * b - 4 * c;

		if (d >= 0)
		{
			float sqrt_d = sqrtf(d);
			float inv2 = 0.5f;
			float step1 = (-b - sqrt_d) * inv2;
			float step2 = (-b + sqrt_d) * inv2;
			// whole wphere is behind camera

			bool var_a = step1 < 0 && step2 >= 0;
			bool var_b = step1 >= 0 && step1 < res.z;
			res.z = var_a ? FLT_MAX : var_b ? step1 : res.z;
			res.index = var_a ? -1 : var_b ? i : res.index;

			//if (step1 >= 0 && step1 < res.z)
			//{
			//	res.z = step1;
			//	res.index = i;
			//}
			//// camera is inside a sphere
			//else if (step1 < 0 && step2 >= 0)
			//{
			//	res.z = FLT_MAX;
			//	res.index = -1;
			//	break;
			//}
		}
		i++;
		//dtmp = dtmp + 1 == n ? 0 : dtmp + 1;
	}
	return res;
}


__device__ HitObj find_intersection_gpu_ver3(float ray_x, float ray_y, Spheres spheres, unsigned char* array, int n, float3 camera_pos, int num)
{
	HitObj res;
	res.x = 0;
	res.y = 0;
	res.z = FLT_MAX;
	res.index = -1;
	float ray_z = -1;
	float radius;
	float3 A = make_float3(ray_x, ray_y, 0);
	// Now hardcode the camera position as 0,0,-500
	float3 B = normalize(A - camera_pos);
	float3 C;
	// I have unit vector for B, so a is 1
	float a = 1.0f;
	float b;
	float c;
	float d;
	float step1;
	float step2;
	for (int i = 0; i < n; i++)
	{
		if (array[i])
		{
			//printf("%d\n", array[i]);
			C = make_float3(spheres.x[i], spheres.y[i], spheres.z[i]);
			radius = spheres.radius[i];
			float3 A_C = A - C;
			b = 2 * dot(B, A_C);
			float tmp = dot(A_C, A_C);
			c = dot(A_C, A_C) - radius * radius;
			d = b * b - 4 * c;
			if (d >= 0)
			{
				float sqrt_d = sqrtf(d);
				float inv2 = 0.5f;
				float step1 = (-b - sqrt_d) * inv2;
				float step2 = (-b + sqrt_d) * inv2;
				// whole wphere is behind camera

				bool var_a = step1 < 0 && step2 >= 0;
				bool var_b = step1 >= 0 && step1 < res.z;
				res.z = var_a ? FLT_MAX : var_b ? step1 : res.z;
				res.index = var_a ? -1 : var_b ? i : res.index;
			}
		}
	}
	return res;
}



__device__ HitObj find_intersection_gpu(float ray_x, float ray_y, float* x, float* y, float* z, float* radiuses, int n, float3 camera_pos)
{
	HitObj res;
	res.x = 0;
	res.y = 0;
	res.z = FLT_MAX;
	res.index = -1;
	float ray_z = -1;
	float radius;
	float3 A = make_float3(ray_x, ray_y, 0);
	// Now hardcode the camera position as 0,0,-500
	float3 B = normalize(A - camera_pos);
	float3 C;
	// I have unit vector for B, so a is 1
	float a = 1.0f;
	float b;
	float c;
	float d;
	float step1;
	float step2;
	for (int i = 0; i < n; i++)
	{
		C = make_float3(x[i], y[i], z[i]);
		//if (ray_x == -199 && ray_y == 200)
		//	printf("%f %f %f\n%f %f %f\n %f %f %f\n", A.x, A.y, A.z, B.x, B.y, B.z, C.x, C.y, C.z);
		radius = radiuses[i];
		float3 A_C = A - C;
		b = 2 * dot(B, A_C);
		c = dot(A_C, A_C) - radius * radius;
		d = b * b - 4 * c;

		//if (ray_x == -199 && ray_y == 200)
		//	printf("%f %f %f %f\n", b, c, d, tmp);
		if (d >= 0)
		{
			step1 = (-b - sqrt(d)) / 2;
			step2 = (-b + sqrt(d)) / 2;
			// whole wphere is behind camera
			if (step1 >= 0 && step1 < res.z)
			{
				res.z = step1;
				res.index = i;
			}
			// camera is inside a sphere
			else if (step1 < 0 && step2 >= 0)
			{
				res.z = FLT_MAX;
				res.index = -1;
				break;
			}
		}
	}
	return res;
}

__host__ __device__ float3 find_color_for_hit(HitObj hit, Spheres spheres, LightSources lights, int nl, float3* ia, int i, int j)
{
	// If no sphere intersection is detected
	if (hit.index == -1)
		return make_float3(0, 0, 0);
	float3 observer_pos = make_float3(i, j, 0);
	float3 sphere_center = make_float3(spheres.x[hit.index], spheres.y[hit.index], spheres.z[hit.index]);
	float3 sphere_color = make_float3(spheres.R[hit.index], spheres.G[hit.index], spheres.B[hit.index]);
	float3 hit_pos = make_float3(hit.x, hit.y, hit.z);
	// Find normal
	float3 N = normalize(hit_pos - sphere_center);
	// Find vector to observer
	float3 V = normalize(observer_pos - hit_pos);
	// For each Light Source find vector to light
	float3 light_pos = make_float3(0, 0, 0);
	float3 light_color;
	float3 L;

	float3 R;
	float LN_dot_prod;
	float RV_dot_prod;
	float kd = spheres.kd[hit.index];
	float ks = spheres.ks[hit.index];
	float alpha = spheres.alpha[hit.index];
	float3 color_of_pixel = make_float3(0, 0, 0);
	for (int k = 0; k < nl; k++)
	{
		light_pos = make_float3(lights.x[k], lights.y[k], lights.z[k]);
		light_color = make_float3(lights.R[k], lights.G[k], lights.B[k]);

		L = normalize(light_pos - hit_pos);

		// Also here find R vector
		R = normalize(2 * dot(L, N) * N - L);

		LN_dot_prod = dot(L, N);
		RV_dot_prod = dot(R, V);

		LN_dot_prod = LN_dot_prod >= 0 ? LN_dot_prod : 0;
		RV_dot_prod = RV_dot_prod >= 0 ? RV_dot_prod : 0;

		color_of_pixel += kd * LN_dot_prod * sphere_color + ks * pow(RV_dot_prod, alpha) * light_color;
	}

	color_of_pixel += spheres.ka[hit.index] * (*ia);
	color_of_pixel = clamp(color_of_pixel, make_float3(0, 0, 0), make_float3(1, 1, 1));
	return color_of_pixel;
}

__device__ void check_if_sphere_is_visible_for_block(
	int x_min, int y_max, int x_max, int y_min,
	float x, float y, float z, float radius,
	unsigned char* array, int index, float3 camera_pos)
{
	
	if (z - radius <= camera_pos.z) {
		array[index] = 0; // Not visible
		return;
	}

	// Compute perspective projection
	float dz = z - camera_pos.z; // Distance to the camera
	


	float proj_x_min = (x - radius) * fabs(camera_pos.z) / dz;
	float proj_x_max = (x + radius) * fabs(camera_pos.z) / dz;
	float proj_y_min = (y - radius) * fabs(camera_pos.z) / dz;
	float proj_y_max = (y + radius) * fabs(camera_pos.z) / dz;

	proj_x_min = min(proj_x_max, proj_x_min);
	proj_y_min = min(proj_y_max, proj_y_min);

	bool x_overlap = !(proj_x_max <= x_min || proj_x_min >= x_max);
	bool y_overlap = !(proj_y_max <= y_min || proj_y_min >= y_max);

	bool x_containing = (proj_x_min <= x_min && proj_x_max >= x_max);
	bool y_containing = (proj_y_min <= y_min && proj_y_max >= y_max);

	array[index] = (x_overlap && y_overlap) || (x_containing && y_overlap) ||
		(x_overlap && y_containing) || (x_containing && y_containing) ? 1 : 0;

	// Debug output
	//printf("Index: %d, Visible: %d, ProjX: [%f, %f], ProjY: [%f, %f], ScreenX: [%d, %d], ScreenY: [%d, %d]\n",
	//	index, array[index], proj_x_min, proj_x_max, proj_y_min, proj_y_max, x_min, x_max, y_min, y_max);
}
