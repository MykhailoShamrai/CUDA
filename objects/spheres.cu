#include "spheres.cuh"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "../includes/cuda_helper.cuh"

void h_allocate_memory_for_spheres(Spheres* spheres, int n)
{
	spheres->x_unrotated = (float*)malloc(sizeof(float) * n);
	spheres->y_unrotated = (float*)malloc(sizeof(float) * n);
	spheres->z_unrotated = (float*)malloc(sizeof(float) * n);
	spheres->x = (float*)malloc(sizeof(float) * n);
	spheres->y = (float*)malloc(sizeof(float) * n);
	spheres->z = (float*)malloc(sizeof(float) * n);
	spheres->radius = (float*)malloc(sizeof(float) * n);
	spheres->R = (float*)malloc(sizeof(float) * n);
	spheres->G = (float*)malloc(sizeof(float) * n);
	spheres->B = (float*)malloc(sizeof(float) * n);
	spheres->ka = (float*)malloc(sizeof(float) * n);
	spheres->kd = (float*)malloc(sizeof(float) * n);
	spheres->ks = (float*)malloc(sizeof(float) * n);
	spheres->alpha = (float*)malloc(sizeof(float) * n);
}

void h_clean_memory_for_spheres(Spheres* spheres)
{
	free(spheres->x_unrotated);
	free(spheres->y_unrotated);
	free(spheres->z_unrotated);
	free(spheres->x);
	free(spheres->y);
	free(spheres->z);
	free(spheres->radius);
	free(spheres->R); 
	free(spheres->G); 
	free(spheres->B); 
	free(spheres->ka);
	free(spheres->kd);
	free(spheres->ks);
	free(spheres->alpha);
}

void d_allocate_memory_for_spheres(Spheres* spheres, int n)
{
	checkCudaErrors(hipMalloc((void**)&spheres->x_unrotated, sizeof(float) * n));
	checkCudaErrors(hipMalloc((void**)&spheres->y_unrotated, sizeof(float) * n));
	checkCudaErrors(hipMalloc((void**)&spheres->z_unrotated, sizeof(float) * n));
	checkCudaErrors(hipMalloc((void**)&(spheres->x), sizeof(float) * n));
	checkCudaErrors(hipMalloc((void**)&(spheres->y), sizeof(float) * n));
	checkCudaErrors(hipMalloc((void**)&(spheres->z), sizeof(float) * n));
	checkCudaErrors(hipMalloc((void**)&(spheres->radius), sizeof(float) * n));
	checkCudaErrors(hipMalloc((void**)&(spheres->R), sizeof(float) * n));
	checkCudaErrors(hipMalloc((void**)&(spheres->G), sizeof(float) * n));
	checkCudaErrors(hipMalloc((void**)&(spheres->B), sizeof(float) * n));
	checkCudaErrors(hipMalloc((void**)&(spheres->ka), sizeof(float) * n));
	checkCudaErrors(hipMalloc((void**)&(spheres->kd), sizeof(float) * n));
	checkCudaErrors(hipMalloc((void**)&(spheres->ks), sizeof(float) * n));
	checkCudaErrors(hipMalloc((void**)&(spheres->alpha), sizeof(float) * n));
}

void d_clean_memory_for_spheres(Spheres* spheres)
{
	checkCudaErrors(hipFree(spheres->x_unrotated));
	checkCudaErrors(hipFree(spheres->y_unrotated));
	checkCudaErrors(hipFree(spheres->z_unrotated));
	checkCudaErrors(hipFree(spheres->x));
	checkCudaErrors(hipFree(spheres->y));
	checkCudaErrors(hipFree(spheres->z));
	checkCudaErrors(hipFree(spheres->radius));
	checkCudaErrors(hipFree(spheres->R));
	checkCudaErrors(hipFree(spheres->G));
	checkCudaErrors(hipFree(spheres->B));
	checkCudaErrors(hipFree(spheres->ka));
	checkCudaErrors(hipFree(spheres->kd));
	checkCudaErrors(hipFree(spheres->ks));
	checkCudaErrors(hipFree(spheres->alpha));
}

void create_random_spheres(Spheres* spheres, int n)
{
	for (int i = 0; i < n; i++)
	{
		spheres->x[i] = rand_float(-1000, 1000);
		spheres->y[i] = rand_float(-1000, 1000);
		spheres->z[i] = rand_float(-1000, 1000);
		spheres->x_unrotated[i] = spheres->x[i];
		spheres->y_unrotated[i] = spheres->y[i];
		spheres->z_unrotated[i] = spheres->z[i];
		spheres->radius[i] = rand_float(10, 20);
		spheres->R[i] = rand_float(0, 1);
		spheres->G[i] = rand_float(0, 1);
		spheres->B[i] = rand_float(0, 1);
		spheres->ka[i] = rand_float(0, 0.2);
		spheres->kd[i] = rand_float(0, 0.5);
		spheres->ks[i] = rand_float(0, 1);
		spheres->alpha[i] = 100;
	}
}