#include "hip/hip_runtime.h"

#include <GLFW/glfw3.h>
#include "../includes/cuda_helper.cuh"
#include "../renderers/kernel_renderer.cuh"
#include "../renderers/cpu_renderer.cuh"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "camera_helpers.cuh"
#include <stdio.h>
#include <cuda_gl_interop.h>


#define NUMBER_OF_SPHERES 10
#define NUMBER_OF_LIGHTS 10
//#define WIDTH 1600
//#define HEIGHT 800
#define THREAD_NUMBER 16
#define THREAD_ROTATE_NUMBER 128

#define SENSETIVITY_OF_MOUSE 0.05f

static bool IS_ANIMATED = true;

static int old_width = 640;
static int old_height = 480;
static int n_width = old_width;
static int n_height = old_height;

static float lastX = old_width / 2.0;
static float lastY = old_height / 2.0;
static bool dragging = false;
static bool light_rotation = false;

static float angle_y_spheres = 0.0f;
static float angle_x_spheres = 0.0f;
static float angle_y_lights = 0.0f;
static float angle_x_lights = 0.0f;

static bool gpu_render = false;

char output_text_buffer[256];

static void animation_callback(GLFWwindow* window, int key, int scancode, int action, int mods)
{
    if (key == GLFW_KEY_SPACE && action == GLFW_PRESS)
    {
        IS_ANIMATED = !IS_ANIMATED;
    }
    else if (key == GLFW_KEY_LEFT_SHIFT && action == GLFW_PRESS)
    {
        light_rotation = !light_rotation;
    }
}

static void mouse_button_callback(GLFWwindow* window, int button, int action, int mods)
{
    if (button == GLFW_MOUSE_BUTTON_LEFT)
    {
        if (action == GLFW_PRESS)
        {
            double xpos, ypos;
            glfwGetCursorPos(window, &xpos, &ypos);
            lastX = xpos;
            lastY = ypos;
            dragging = true;
            
        }
        else if (action == GLFW_RELEASE)
        {
            dragging = false;
        }
    }
}

static void cursor_position_callback(GLFWwindow* window, double xpos, double ypos)
{
    if (dragging)
    {
        // Calculate mouse movement offsets
        float xoffset = lastX - xpos;
        float yoffset = lastY - ypos;
        lastX = xpos;
        lastY = ypos;
    
        xoffset *= SENSETIVITY_OF_MOUSE;
        yoffset *= SENSETIVITY_OF_MOUSE;

        if (light_rotation)
        {
            angle_x_lights += xoffset;
            angle_y_lights += yoffset;
        }
        else
        {
            angle_x_spheres += xoffset;
            angle_y_spheres += yoffset;
        }
    }
}

static void framebuffer_size_callback(GLFWwindow*, int new_width, int new_height)
{
    n_width = new_width;
    n_height = new_height;
    glViewport(0, 0, new_width, new_height);
}


int main(void)
{
    Spheres spheres;
    Spheres d_spheres;
    LightSources lights;
    LightSources d_lights;

    h_allocate_memory_for_spheres(&spheres, NUMBER_OF_SPHERES);
    create_random_spheres(&spheres, NUMBER_OF_SPHERES);

    h_allocate_memory_for_light_sources(&lights, NUMBER_OF_LIGHTS);
    create_random_light_sources(&lights, NUMBER_OF_LIGHTS);
   
    d_allocate_memory_for_spheres(&d_spheres, NUMBER_OF_SPHERES);
    d_allocate_memory_for_light_sources(&d_lights, NUMBER_OF_LIGHTS);

    float* unrotated_x_lights = (float*)malloc(sizeof(float) * NUMBER_OF_LIGHTS);
    float* unrotated_y_lights = (float*)malloc(sizeof(float) * NUMBER_OF_LIGHTS);
    float* unrotated_z_lights = (float*)malloc(sizeof(float) * NUMBER_OF_LIGHTS);
    for (int i = 0; i < NUMBER_OF_LIGHTS; i++)
    {
        unrotated_x_lights[i] = lights.x[i];
        unrotated_y_lights[i] = lights.y[i];
        unrotated_z_lights[i] = lights.z[i];
    }


    checkCudaErrors(hipMemcpy(d_spheres.x_unrotated, spheres.x_unrotated, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.y_unrotated, spheres.y_unrotated, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.z_unrotated, spheres.z_unrotated, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.x, spheres.x, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.y, spheres.y, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.z, spheres.z, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.ka, spheres.ka, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.ks, spheres.ks, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.kd, spheres.kd, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.R, spheres.R, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.G, spheres.G, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.B, spheres.B, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.alpha, spheres.alpha, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.radius, spheres.radius, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));


    checkCudaErrors(hipMemcpy(d_lights.x_unrotated, lights.x_unrotated, NUMBER_OF_LIGHTS * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_lights.y_unrotated, lights.y_unrotated, NUMBER_OF_LIGHTS * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_lights.z_unrotated, lights.z_unrotated, NUMBER_OF_LIGHTS * sizeof(float), hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(d_lights.x, lights.x, NUMBER_OF_LIGHTS * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_lights.y, lights.y, NUMBER_OF_LIGHTS * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_lights.z, lights.z, NUMBER_OF_LIGHTS * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_lights.R, lights.R, NUMBER_OF_LIGHTS * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_lights.G, lights.G, NUMBER_OF_LIGHTS * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_lights.B, lights.B, NUMBER_OF_LIGHTS * sizeof(float), hipMemcpyHostToDevice));

    float3 camera_pos = make_float3(0, 0, - n_width / 2);


    float* h_bitmap = (float*)malloc(n_width * n_height * 3 * sizeof(float));
    float* d_bitmap;
    checkCudaErrors(hipMalloc((void**)&d_bitmap, n_width * n_height * 3 * sizeof(float)));

    glfwWindowHint(GLFW_RESIZABLE, GLFW_TRUE);
    if (!glfwInit())
        return -1;

    /* Create a windowed mode window and its OpenGL context */
    GLFWwindow* window = glfwCreateWindow(n_width, n_height, "Test Window", NULL, NULL);
    if (!window)
    {
        glfwTerminate();
        return -1;
    }

    glfwMakeContextCurrent(window);
    glViewport(0, 0, n_width, n_height);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0, n_width, 0, n_height, -1, 1);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    glGetString(GL_VERSION);

    glfwSetKeyCallback(window, animation_callback);
    glfwSetMouseButtonCallback(window, mouse_button_callback);
    glfwSetCursorPosCallback(window, cursor_position_callback);
    glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);

    

    // Initialisation of timers
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEvent_t start_mem, stop_mem;
    hipEventCreate(&start_mem);
    hipEventCreate(&stop_mem);
    hipEvent_t start_rotate, stop_rotate;
    hipEventCreate(&start_rotate);
    hipEventCreate(&stop_rotate);

    int dim_blocks_x = (n_width + THREAD_NUMBER - 1) / THREAD_NUMBER;
    int dim_blocks_y = (n_height + THREAD_NUMBER - 1) / THREAD_NUMBER;


    // block for framecounting
    int number_of_frames = 0;
    double last_time = glfwGetTime();
    double last_time_anim = glfwGetTime();
    int frame_rate = 0;

    dim3 blocks(dim_blocks_x, dim_blocks_y);
    dim3 threads(THREAD_NUMBER, THREAD_NUMBER);

    int blocks_for_rotation = (NUMBER_OF_SPHERES + THREAD_ROTATE_NUMBER - 1) / THREAD_ROTATE_NUMBER;

    //float3 new_camera_pos = camera_pos;
    while (!glfwWindowShouldClose(window))
    {
       
        number_of_frames++;
        double current_time = glfwGetTime();
        if (IS_ANIMATED)
        {
            double time_diff = current_time - last_time_anim;
            angle_x_spheres += time_diff * 3.0f;
            angle_y_spheres += time_diff * 3.0f;
        }
        last_time_anim = current_time;

        if (current_time - last_time >= 1.0)
        {
            frame_rate = number_of_frames;
            number_of_frames = 0;
            last_time += 1.0;
        }


        // Important for resizing a window
        if (n_width != old_width || n_height != old_height)
        {
            free(h_bitmap);
            checkCudaErrors(hipFree(d_bitmap));
            h_bitmap = (float*)malloc(n_width * n_height * 3 * sizeof(float));
            checkCudaErrors(hipMalloc((void**)&d_bitmap, n_width * n_height * 3 * sizeof(float)));
            old_height = n_height;
            old_width = n_width;
            camera_pos = make_float3(0, 0, -n_width / 2);
            dim_blocks_x = (n_width + THREAD_NUMBER - 1) / THREAD_NUMBER;
            dim_blocks_y = (n_height + THREAD_NUMBER - 1) / THREAD_NUMBER;
            blocks = dim3(dim_blocks_x, dim_blocks_y);
        }

        // Rotation

        angle_x_spheres = angle_x_spheres > 360.0f ? 0 : angle_x_spheres < -360.0f ? 0 : angle_x_spheres;
        angle_y_spheres = angle_y_spheres > 360.0f ? 0 : angle_y_spheres < -360.0f ? 0 : angle_y_spheres;
        angle_x_lights = angle_x_lights > 360.0f ? 0 : angle_x_lights < -360.0f ? 0 : angle_x_lights;
        angle_y_lights = angle_y_lights > 360.0f ? 0 : angle_y_lights < -360.0f ? 0 : angle_y_lights;

        // CPU PART
            // Rotate
            // DRAW


        // CPU PART

        float elapsed_time = 0;
        float elapsed_time_mem = 0;
        float elapsed_time_rotation = 0;
        // KERNEL PART
        if (gpu_render)
        {
            hipEventRecord(start_rotate);
            rotate_objects << <blocks_for_rotation, THREAD_ROTATE_NUMBER >> > (d_spheres, d_lights, angle_x_spheres, angle_y_spheres, angle_x_lights, angle_y_lights,
                NUMBER_OF_SPHERES, NUMBER_OF_LIGHTS);
            checkCudaErrors(hipGetLastError());
            checkCudaErrors(hipDeviceSynchronize());
            hipEventRecord(stop_rotate);
            hipEventRecord(start);
            
            unsigned shmem_size = sizeof(unsigned char) * NUMBER_OF_SPHERES;
            refresh_bitmap << <blocks, threads, shmem_size >> > (d_bitmap, d_spheres, NUMBER_OF_SPHERES, d_lights, NUMBER_OF_LIGHTS,
                n_width, n_height, camera_pos);
            checkCudaErrors(hipGetLastError());
            checkCudaErrors(hipDeviceSynchronize());
            hipEventRecord(stop);
            
            hipEventRecord(start_mem);
            checkCudaErrors(hipMemcpy(h_bitmap, d_bitmap, n_width * n_height * 3 * sizeof(float), hipMemcpyDeviceToHost));
            hipEventRecord(stop_mem);
            hipEventElapsedTime(&elapsed_time, start, stop);
            hipEventElapsedTime(&elapsed_time_mem, start_mem, stop_mem);
            hipEventElapsedTime(&elapsed_time_rotation, start_rotate, stop_rotate);
           
        }
        else
        {
            //free(h_bitmap);
            //h_bitmap = (float*)malloc(sizeof(float) * 3 * n_width * n_height);
            double start_rot = glfwGetTime();
            for (int i = 0; i < NUMBER_OF_SPHERES; i++)
            {
                rotate_positions(&spheres.x[i], &spheres.z[i], &spheres.x_unrotated[i], &spheres.z_unrotated[i], angle_x_spheres);
                rotate_positions(&spheres.y[i], &spheres.z[i], &spheres.y_unrotated[i], &spheres.z[i], angle_y_spheres);
            }
            double end_rot = glfwGetTime();
            elapsed_time_rotation = end_rot - start_rot;

            refresh_bitmap_cpu(h_bitmap, spheres, NUMBER_OF_SPHERES, lights, NUMBER_OF_LIGHTS, n_width,
                n_height, camera_pos);
        }
        // KERNEL PART
        
        sprintf(output_text_buffer, "FPS: %d :: TIME FOR MEMORY COPY %f :: TIME FOR KERNEL EXECUTION :: %f :: ROTATION %f", frame_rate, elapsed_time_mem, elapsed_time, elapsed_time_rotation);
        
        glfwSetWindowTitle(window, output_text_buffer);
        glClear(GL_COLOR_BUFFER_BIT);

        
        glDrawPixels(n_width, n_height, GL_RGB, GL_FLOAT, h_bitmap);
        glfwSwapBuffers(window);

        glfwPollEvents();

    }
    glfwTerminate();
    
    

    // cleaning 
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(start_mem);
    hipEventDestroy(stop_mem);

    free(h_bitmap);
    checkCudaErrors(hipFree(d_bitmap));
    d_clean_memory_for_spheres(&d_spheres);
    h_clean_memory_for_light_sources(&lights);
    d_clean_memory_for_light_sources(&d_lights);
    h_clean_memory_for_spheres(&spheres);
    return 0;
}