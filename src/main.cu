#include "hip/hip_runtime.h"

#include <GLFW/glfw3.h>
#include "../includes/cuda_helper.cuh"
#include "../renderers/kernel_renderer.cuh"
#include "../renderers/cpu_renderer.cuh"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "camera_helpers.cuh"


#define NUMBER_OF_SPHERES 500
#define NUMBER_OF_LIGHTS 100
#define WIDTH 1600
#define HEIGHT 900

#define THREAD_NUMBER 16


int main(void)
{
    Spheres spheres;
    Spheres d_spheres;
    LightSources lights;
    LightSources d_lights;
    
    h_allocate_memory_for_spheres(&spheres, NUMBER_OF_SPHERES);
    //create_test_spheres(&spheres);
    create_random_spheres(&spheres, NUMBER_OF_SPHERES);

    h_allocate_memory_for_light_sources(&lights, NUMBER_OF_LIGHTS);
    create_random_light_sources(&lights, NUMBER_OF_LIGHTS);
    

    d_allocate_memory_for_spheres(&d_spheres, NUMBER_OF_SPHERES);
    d_allocate_memory_for_light_sources(&d_lights, NUMBER_OF_LIGHTS);

    checkCudaErrors(hipMemcpy(d_spheres.x, spheres.x, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.y, spheres.y, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.z, spheres.z, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.ka, spheres.ka, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.ks, spheres.ks, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.kd, spheres.kd, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.R, spheres.R, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.G, spheres.G, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.B, spheres.B, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.alpha, spheres.alpha, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.radius, spheres.radius, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(d_lights.x, lights.x, NUMBER_OF_LIGHTS * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_lights.y, lights.y, NUMBER_OF_LIGHTS * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_lights.z, lights.z, NUMBER_OF_LIGHTS * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_lights.R, lights.R, NUMBER_OF_LIGHTS * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_lights.G, lights.G, NUMBER_OF_LIGHTS * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_lights.B, lights.B, NUMBER_OF_LIGHTS * sizeof(float), hipMemcpyHostToDevice));

    float3 camera_pos = make_float3(0, 0, - WIDTH / 2);


    unsigned char* h_bitmap = (unsigned char*)malloc(WIDTH * HEIGHT * 3  * sizeof(unsigned char));
    unsigned char* d_bitmap;
    checkCudaErrors(hipMalloc((void**)&d_bitmap, WIDTH * HEIGHT * 3 * sizeof(unsigned char)));

    int dim_blocks_x = (WIDTH + THREAD_NUMBER - 1) / THREAD_NUMBER;
    int dim_blocks_y = (HEIGHT + THREAD_NUMBER - 1) / THREAD_NUMBER;

    dim3 blocks(dim_blocks_x, dim_blocks_y);
    dim3 threads(THREAD_NUMBER, THREAD_NUMBER);




    //refresh_bitmap_cpu(h_bitmap, spheres, NUMBER_OF_SPHERES, lights, NUMBER_OF_LIGHTS, WIDTH, HEIGHT, camera_pos);


    /* Initialize the library */
    if (!glfwInit())
        return -1;

    /* Create a windowed mode window and its OpenGL context */
    GLFWwindow* window = glfwCreateWindow(WIDTH, HEIGHT, "Test Window", NULL, NULL);
    if (!window) {
        glfwTerminate();
        return -1;
    }

    /* Make the window's context current */
    glfwMakeContextCurrent(window);
    glViewport(0, 0, WIDTH, HEIGHT);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0, WIDTH, 0, HEIGHT, -1, 1);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    //refresh_bitmap << <blocks, threads >> > (d_bitmap, d_spheres, NUMBER_OF_SPHERES, d_lights, NUMBER_OF_LIGHTS, WIDTH, HEIGHT, camera_pos);
    //checkCudaErrors(hipGetLastError());
    ////hipDeviceSynchronize();
    //checkCudaErrors(hipDeviceSynchronize());
    //
    //checkCudaErrors(hipMemcpy(h_bitmap, d_bitmap, WIDTH * HEIGHT * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    hipEvent_t start_mem, stop_mem;
    hipEventCreate(&start_mem);
    hipEventCreate(&stop_mem);


    float3 new_camera_pos = camera_pos;
    int angle = 0;
    /* Loop until the user closes the window */
    while (!glfwWindowShouldClose(window))
    {
        hipEventRecord(start);
        unsigned shmem_size = sizeof(float) * 4 * NUMBER_OF_SPHERES;
        refresh_bitmap << <blocks, threads, shmem_size>> > (d_bitmap, d_spheres, NUMBER_OF_SPHERES, d_lights, NUMBER_OF_LIGHTS, WIDTH, HEIGHT, camera_pos);
        checkCudaErrors(hipGetLastError());
        //hipDeviceSynchronize();

        checkCudaErrors(hipDeviceSynchronize());
        //
        hipEventRecord(stop);


        hipEventRecord(start_mem);
        checkCudaErrors(hipMemcpy(h_bitmap, d_bitmap, WIDTH * HEIGHT * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost));
        hipEventRecord(stop_mem);

        float elapsed_time;
        hipEventElapsedTime(&elapsed_time, start, stop);
        printf("time for generation of frame: %f\n", elapsed_time);
        hipEventElapsedTime(&elapsed_time, start_mem, stop_mem);
        printf("time for memory copying: %f\n", elapsed_time);

        /* Render here */
        glClear(GL_COLOR_BUFFER_BIT);

        /* Swap front and back buffers */
        glDrawPixels(WIDTH, HEIGHT, GL_RGB, GL_UNSIGNED_BYTE, h_bitmap);
        glfwSwapBuffers(window);

        /* Poll for and process events */
        glfwPollEvents();
    }
    glfwTerminate();
    
    

    // cleaning 
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(start_mem);
    hipEventDestroy(stop_mem);

    free(h_bitmap);
    checkCudaErrors(hipFree(d_bitmap));
    d_clean_memory_for_spheres(&d_spheres);
    h_clean_memory_for_light_sources(&lights);
    d_clean_memory_for_light_sources(&d_lights);
    h_clean_memory_for_spheres(&spheres);
    return 0;
}