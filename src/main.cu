
#include <GLFW/glfw3.h>
//#include "../objects/spheres.cuh"
#include "../includes/cuda_helper.cuh"
#include "../renderers/kernel_renderer.cuh"
#include "../renderers/cpu_renderer.cuh"
#include <stdlib.h>
#include <hip/hip_runtime.h>

//#include "cmake-test-cuda.h"

#define NUMBER_OF_SPHERES 10
#define NUMBER_OF_LIGHTS 10
#define WIDTH 1200
#define HEIGHT 900

#define THREAD_NUMBER 16


int main(void)
{
    Spheres sphere;
    Spheres d_spheres;
    LightSources lights;
    LightSources d_lights;

    
    
    //hipMalloc((void**)&d_spheres, sizeof(Spheres));
    //hipMalloc((void**)&d_lights, sizeof(LightSources));
    //
    //
    //checkCudaErrors(hipMemcpy(&d_lights, &lights, sizeof(LightSources), hipMemcpyHostToDevice));
    //checkCudaErrors(hipMemcpy(&d_spheres, &sphere, sizeof(LightSources), hipMemcpyHostToDevice));


    
    h_allocate_memory_for_spheres(&sphere, NUMBER_OF_SPHERES);
    //create_test_spheres(&sphere);
    create_random_spheres(&sphere, NUMBER_OF_SPHERES);

    h_allocate_memory_for_light_sources(&lights, NUMBER_OF_LIGHTS);
    create_random_light_sources(&lights, NUMBER_OF_LIGHTS);
    

    d_allocate_memory_for_spheres(&d_spheres, NUMBER_OF_SPHERES);
    d_allocate_memory_for_light_sources(&d_lights, NUMBER_OF_LIGHTS);

    checkCudaErrors(hipMemcpy(d_spheres.x, sphere.x, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.y, sphere.y, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.z, sphere.z, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.ka, sphere.ka, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.ks, sphere.ks, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.kd, sphere.kd, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.R, sphere.R, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.G, sphere.G, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.B, sphere.B, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_spheres.alpha, sphere.alpha, NUMBER_OF_SPHERES * sizeof(float), hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(d_lights.x, lights.x, NUMBER_OF_LIGHTS * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_lights.y, lights.y, NUMBER_OF_LIGHTS * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_lights.z, lights.z, NUMBER_OF_LIGHTS * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_lights.R, lights.R, NUMBER_OF_LIGHTS * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_lights.G, lights.G, NUMBER_OF_LIGHTS * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_lights.B, lights.B, NUMBER_OF_LIGHTS * sizeof(float), hipMemcpyHostToDevice));



    GLubyte* h_bitmap = (GLubyte*)malloc(WIDTH * HEIGHT * 3  * sizeof(GLubyte));
    GLubyte* d_bitmap;
    checkCudaErrors(hipMalloc((void**)&d_bitmap, WIDTH * HEIGHT * 3 * sizeof(GLubyte)));

    int dim_blocks_x = (WIDTH + THREAD_NUMBER - 1) / THREAD_NUMBER;
    int dim_blocks_y = (HEIGHT + THREAD_NUMBER - 1) / THREAD_NUMBER;

    dim3 blocks(dim_blocks_x, dim_blocks_y);
    dim3 threads(THREAD_NUMBER, THREAD_NUMBER);

    refresh_bitmap << <blocks, threads >> > (d_bitmap, d_spheres, NUMBER_OF_SPHERES, d_lights, NUMBER_OF_LIGHTS, WIDTH, HEIGHT);
    checkCudaErrors(hipGetLastError());
    hipDeviceSynchronize();
    //checkCudaErrors(hipDeviceSynchronize());
    
    checkCudaErrors(hipMemcpy(h_bitmap, d_bitmap, WIDTH * HEIGHT * 3 * sizeof(GLubyte), hipMemcpyDeviceToHost));


    //refresh_bitmap_cpu(h_bitmap, sphere, NUMBER_OF_SPHERES, lights, NUMBER_OF_LIGHTS, WIDTH, HEIGHT);


    /* Initialize the library */
    if (!glfwInit())
        return -1;

    /* Create a windowed mode window and its OpenGL context */
    GLFWwindow* window = glfwCreateWindow(WIDTH, HEIGHT, "Test Window", NULL, NULL);
    if (!window) {
        glfwTerminate();
        return -1;
    }

    /* Make the window's context current */
    glfwMakeContextCurrent(window);
    glViewport(0, 0, WIDTH, HEIGHT);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0, WIDTH, 0, HEIGHT, -1, 1);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();




    /* Loop until the user closes the window */
    while (!glfwWindowShouldClose(window))
    {
        /* Render here */
        glClear(GL_COLOR_BUFFER_BIT);

        /* Swap front and back buffers */
        glDrawPixels(WIDTH, HEIGHT, GL_RGB, GL_UNSIGNED_BYTE, h_bitmap);
        glfwSwapBuffers(window);

        /* Poll for and process events */
        glfwPollEvents();
    }
    glfwTerminate();
    
    

    free(h_bitmap);
    checkCudaErrors(hipFree(d_bitmap));
    d_clean_memory_for_spheres(&d_spheres);
    h_clean_memory_for_light_sources(&lights);
    d_clean_memory_for_light_sources(&d_lights);
    h_clean_memory_for_spheres(&sphere);
    //checkCudaErrors(hipFree(&d_spheres));
    //checkCudaErrors(hipFree(&d_lights));
    return 0;
}